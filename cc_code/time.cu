#include "hip/hip_runtime.h"
//这个代码的目的是比较cuda并行计算与普通cpu串行计算速度，在N=5000的时候，cuda需要1s，而cpu需要33s.随着N增大，这个差别将更加明显。
//当然这并不是一个好的并行计算代码。因为各个block之间的结果是互相影响的，这会导致并不能得到准确结果。

#include "../common/book.h"
#include<time.h>

#define N   5000


/*
int main()
{
	time_t begin,end; //time for seconds
    begin=time(NULL);
    int a;
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<i;j++)
        {
            for(int k=0;k<j;k++)
            {
                a=j;
            }
        }
        
    }
    printf("%d\n",a);
    end=time(NULL);
	printf("use time %d s\n",int(end-begin));
	return 0;
}
*/
//*
__global__ void cc_copy( int *b, int *c ) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
    {
        for(int j=0;j<b[tid];j++)
        {
            for(int k=0;k<j;k++)
            {
                *c=j;
            }
        }
    }
}


int main()
{
	time_t begin,end; //time for seconds
    begin=time(NULL);
    int b[N];
    for (int i=0; i<N; i++) {
        b[i] = i;
    }
    int *dev_b;
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice ) );
    int c;
    int* dev_c;
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    cc_copy<<<N,1>>>( dev_b,dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost ) );

    printf("%d\n",c);
    end=time(NULL);
	printf("use time %d s\n",int(end-begin));
	return 0;
}
//*/